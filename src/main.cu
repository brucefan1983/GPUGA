#include "hip/hip_runtime.h"
/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


#include "gpuga.cuh"
#include "error.cuh"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>


void print_welcome_information(void);
int get_number_of_input_directories(void);


int main(int argc, char *argv[])
{
    print_welcome_information();
    int number_of_inputs = get_number_of_input_directories();

    for (int n = 0; n < number_of_inputs; ++n)
    {
        char input_directory[200];
        int count = scanf("%s", input_directory);
        if (count != 1)
        {
            print_error("reading error for input directory.\n");
        }

        print_line_1();
        printf("Run simulation for '%s'.\n", input_directory);
        print_line_2();

        clock_t time_begin = clock();
        GPUGA gpuga(input_directory);
        clock_t time_finish = clock();

        float time_used = (time_finish - time_begin) / float(CLOCKS_PER_SEC);

        print_line_1();
        printf("Time used for '%s' = %f s.\n", input_directory, time_used);
        print_line_2();
    }

    print_line_1();
    printf("Finished running gpuga.\n");
    print_line_2();

    return EXIT_SUCCESS;
}


void print_welcome_information(void)
{
    printf("\n");
    printf("***************************************************************\n");
    printf("*                 Welcome to use GPUGA                        *\n");
    printf("*     (Graphics Processing Units Genetic Algorithm)           *\n");
    printf("*                      Version 1.0                            *\n");
    printf("* Author:                                                     *\n");
    printf("*     Zheyong Fan <brucenju@gmail.com>                        *\n");
    printf("***************************************************************\n");
    printf("\n");
}


int get_number_of_input_directories(void)
{
    int number_of_inputs;
    int count = scanf("%d", &number_of_inputs);
    if (count != 1)
    {
        print_error("reading error for number of inputs.\n");
    }
    return number_of_inputs;
}


