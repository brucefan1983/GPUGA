#include "hip/hip_runtime.h"
/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Calculate force, energy, and virial
------------------------------------------------------------------------------*/


#include "potential.cuh"
#include "neighbor.cuh"
#include "mic.cuh"
#include "error.cuh"

const float PI              = 3.141592653589793;

//Easy labels for indexing
const int D0                = 0;
const int A                 = 1;
const int R0                = 2;
const int S                 = 3;
const int EN                = 4;
const int BETA              = 5;
const int H                 = 6;
const int R1                = 7;
const int R2                = 8;
const int PI_FACTOR         = 9;
const int MINUS_HALF_OVER_N = 10;
const int LK                 = 11;



void Potential::initialize(int N, int MAX_ATOM_NUMBER)
{
    int mem_size = sizeof(float) * N * MAX_ATOM_NUMBER;
    CHECK(hipMalloc((void**)&b, mem_size));
    CHECK(hipMalloc((void**)&bp, mem_size));
    CHECK(hipMalloc((void**)&f12x, mem_size));
    CHECK(hipMalloc((void**)&f12y, mem_size));
    CHECK(hipMalloc((void**)&f12z, mem_size));
}


Potential::~Potential(void)
{
    CHECK(hipFree(b));
    CHECK(hipFree(bp));
    CHECK(hipFree(f12x));
    CHECK(hipFree(f12y));
    CHECK(hipFree(f12z));
}


void Potential::update_potential(float* potential_parameters)
{
    pot_para.ters[D0]   = potential_parameters[0];
    pot_para.ters[A]    = potential_parameters[1];
    pot_para.ters[R0]   = potential_parameters[2];
    pot_para.ters[S]    = potential_parameters[3];
    pot_para.ters[EN]   = potential_parameters[4];
    pot_para.ters[BETA] = potential_parameters[5];
    pot_para.ters[H]    = potential_parameters[6];
    pot_para.ters[R1]   = potential_parameters[7];
    pot_para.ters[R2]   = potential_parameters[8];
    pot_para.ters[PI_FACTOR] = PI / (pot_para.ters[R2] - pot_para.ters[R1]);
    pot_para.ters[MINUS_HALF_OVER_N] = - 0.5 / pot_para.ters[EN];
	pot_para.ters[LK]    = potential_parameters[9];
}


static __device__ void find_fr_and_frp
(float d0, float a, float r0, float s, float d12, float &fr, float &frp)
{

    fr = d0 / (s - 1.0f) * exp(-sqrt(2.0f * s) * a * (d12 - r0));
    frp = -sqrt(2.0f * s) * a * fr;
}


static __device__ void find_fa_and_fap
(float d0, float a, float r0, float s, float d12, float &fa, float &fap)
{
    fa = s * d0 / (s - 1.0f) * exp(-sqrt(2.0f / s) * a * (d12 - r0));
    fap = -sqrt(2.0f / s) * a * fa;
}



static __device__ void find_fa
(float d0, float a, float r0, float s, float d12, float &fa)
{
    fa = s * d0 / (s - 1.0f) * exp(-sqrt(2.0f / s) * a * (d12 - r0));
}

static __device__ void find_fc_and_fcp
(float r1, float r2, float pi_factor, float d12, float &fc, float &fcp)
{
    if (d12 < r1) {fc = 1.0f; fcp = 0.0f;}
    else if (d12 < r2)
    {
        fc = 0.5f * cos(pi_factor * (d12 - r1)) + 0.5f;
        fcp = - sin(pi_factor * (d12 - r1)) * pi_factor * 0.5f;
    }
    else {fc  = 0.0f; fcp = 0.0f;}
}


static __device__ void find_fc
(float r1, float r2, float pi_factor, float d12, float &fc)
{
    if (d12 < r1) {fc  = 1.0f;}
    else if (d12 < r2)
    {
        fc = 0.5f * cos(pi_factor * (d12 - r1)) + 0.5f;
    }
    else {fc  = 0.0f;}
}


static __device__ void find_g_and_gp
( float h, float cos, float &g, float &gp)
{
    float x = cos - h;
    g = x * x;
    gp = 2.0f * x;
}


static __device__ void find_g
( float h, float cos, float &g)
{
    float x = cos - h;
    g =  x * x;
}


// 2-body part (kernel)
static __global__ void find_force_tersoff_step0
(
    int number_of_particles, int* Na, int* Na_sum, 
    int* g_neighbor_number, int* g_neighbor_list,
    Pot_Para pot_para,
    const float* __restrict__ g_x,
    const float* __restrict__ g_y,
    const float* __restrict__ g_z,
	const float* __restrict__ g_box,
	float* g_fx, float* g_fy, float* g_fz,
    float* g_virial, float* g_potential
)
{
    int N1 = Na_sum[blockIdx.x];
    int N2 = N1 + Na[blockIdx.x];
    int n1 = N1 + threadIdx.x;
    if (n1 < N2)
    {
   	    float s_fx = 0.0f; // force_x
        float s_fy = 0.0f; // force_y
        float s_fz = 0.0f; // force_z
        float s_pe = 0.0f; // potential energy
        float s_virial_xx = 0.0f; // virial_stress_xx
	    float s_virial_yy = 0.0f; // virial_stress_yy
	    float s_virial_zz = 0.0f; // virial_stress_zz
        float s_virial_xy = 0.0f; // virial_stress_xy
        float s_virial_yz = 0.0f; // virial_stress_yz
        float s_virial_zx = 0.0f; // virial_stress_zx
		const float* __restrict__ h = g_box + 18 * blockIdx.x;		
		int neighbor_number = g_neighbor_number[n1];
        float x1 = LDG(g_x, n1); 
        float y1 = LDG(g_y, n1); 
        float z1 = LDG(g_z, n1);      
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];           
            float x12  = LDG(g_x, n2) - x1;
            float y12  = LDG(g_y, n2) - y1;
            float z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(h, x12, y12, z12);
            float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);          
            float fa12, fap12, fr12, frp12;
            find_fa_and_fap(pot_para.ters[D0], pot_para.ters[A], pot_para.ters[R0], pot_para.ters[S], d12, fa12, fap12);
            find_fr_and_frp(pot_para.ters[D0], pot_para.ters[A], pot_para.ters[R0], pot_para.ters[S], d12, fr12, frp12);
            float p2 = fr12 - fa12;
            float f2 = (frp12 - fap12) / d12;
            
            // treat two-body potential in the same way as many-body potential
			
            float f12x = f2 * x12 * 0.5; 
            float f12y = f2 * y12 * 0.5; 
            float f12z = f2 * z12 * 0.5; 
            float f21x = -f12x; 
            float f21y = -f12y; 
            float f21z = -f12z; 
       
            // accumulate force
            s_fx += f12x - f21x; 
            s_fy += f12y - f21y; 
            s_fz += f12z - f21z; 

            // accumulate potential energy and virial 
            s_pe += p2 * 0.5; // two-body potential
			
            s_virial_xx -= x12 * (f12x - f21x) * 0.5f;
            s_virial_yy -= y12 * (f12y - f21y) * 0.5f;
            s_virial_zz -= z12 * (f12z - f21z) * 0.5f;
            s_virial_xy -= x12 * (f12y - f21y) * 0.5f;
            s_virial_yz -= y12 * (f12z - f21z) * 0.5f;
            s_virial_zx -= z12 * (f12x - f21x) * 0.5f;
           
            
        }

        
        g_fx[n1] = s_fx * pot_para.ters[LK]; // save force
        g_fy[n1] = s_fy * pot_para.ters[LK];
        g_fz[n1] = s_fz * pot_para.ters[LK];

        // save virial
        // xx xy xz    0 3 4
        // yx yy yz    6 1 5
        // zx zy zz    7 8 2
       

        g_virial[n1] = s_virial_xx;
        g_virial[n1 + number_of_particles] = s_virial_yy;
        g_virial[n1 + number_of_particles * 2] = s_virial_zz;
        g_virial[n1 + number_of_particles * 3] = s_virial_xy;
        g_virial[n1 + number_of_particles * 4] = s_virial_yz;
        g_virial[n1 + number_of_particles * 5] = s_virial_zx;
		
        // save potential
        g_potential[n1] = s_pe;
    }
}

// step 1: pre-compute all the bond-order functions and their derivatives
static __global__ void find_force_tersoff_step1
(
    int number_of_particles, int *Na, int *Na_sum, 
    int* g_neighbor_number, int* g_neighbor_list, int* g_type,
    Pot_Para pot_para,
    const float* __restrict__ g_x,
    const float* __restrict__ g_y,
    const float* __restrict__ g_z,
    const float* __restrict__ g_box,
    float* g_b, float* g_bp
)
{
    int N1 = Na_sum[blockIdx.x];
    int N2 = N1 + Na[blockIdx.x];
    int n1 = N1 + threadIdx.x;
    if (n1 < N2)
    {
        const float* __restrict__ h = g_box + 18 * blockIdx.x;
        int neighbor_number = g_neighbor_number[n1];

        float x1 = LDG(g_x, n1); 
        float y1 = LDG(g_y, n1); 
        float z1 = LDG(g_z, n1);
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int n2 = g_neighbor_list[n1 + number_of_particles * i1];

            float x12  = LDG(g_x, n2) - x1;
            float y12  = LDG(g_y, n2) - y1;
            float z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(h, x12, y12, z12);
            float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            float zeta = 0.0f;
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int n3 = g_neighbor_list[n1 + number_of_particles * i2];
                if (n3 == n2) { continue; } // ensure that n3 != n2

                float x13 = LDG(g_x, n3) - x1;
                float y13 = LDG(g_y, n3) - y1;
                float z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(h, x13, y13, z13);
                float d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                float cos123 = (x12 * x13 + y12 * y13 + z12 * z13) / (d12*d13);
                float fc13, g123;
				find_fc
                (
                    pot_para.ters[R1], pot_para.ters[R2], 
                    pot_para.ters[PI_FACTOR], d13, fc13
                );
                find_g(pot_para.ters[H], cos123, g123);
                zeta += fc13 * g123;
            }
            float bzn, b_ijj;
            bzn = pow(pot_para.ters[BETA] *zeta, pot_para.ters[EN]);
            b_ijj = pow(1.0f + bzn, pot_para.ters[MINUS_HALF_OVER_N]);
           
			if (zeta < 1.0e-16) // avoid division by 0
            {
                g_b[i1 * number_of_particles + n1]  = 1.0;
                g_bp[i1 * number_of_particles + n1] = 0.0;
            }
            else
            {
                g_b[i1 * number_of_particles + n1]  = b_ijj;
                g_bp[i1 * number_of_particles + n1] = 
                 - b_ijj * bzn * 0.5f / ((1.0f + bzn) * zeta);
            }
        }
    }
}


// step 2: calculate all the partial forces dU_i/dr_ij
static __global__ void find_force_tersoff_step2
(
    int number_of_particles, int *Na, int *Na_sum, 
    int *g_neighbor_number, int *g_neighbor_list, int *g_type,
    Pot_Para pot_para,
    const float* __restrict__ g_b,
    const float* __restrict__ g_bp,
    const float* __restrict__ g_x,
    const float* __restrict__ g_y,
    const float* __restrict__ g_z,
    const float* __restrict__ g_box,
    float *g_potential, float *g_f12x, float *g_f12y, float *g_f12z
)
{
    int N1 = Na_sum[blockIdx.x];
    int N2 = N1 + Na[blockIdx.x];
    int n1 = N1 + threadIdx.x;
    if (n1 < N2)
    {
        const float* __restrict__ h = g_box + 18 * blockIdx.x;
        int neighbor_number = g_neighbor_number[n1];

        float x1 = LDG(g_x, n1); 
        float y1 = LDG(g_y, n1); 
        float z1 = LDG(g_z, n1);
        float pot_energy = 0.0f;
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * number_of_particles + n1;
            int n2 = g_neighbor_list[index];

            float x12  = LDG(g_x, n2) - x1;
            float y12  = LDG(g_y, n2) - y1;
            float z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(h, x12, y12, z12);
            float d12 = sqrt(x12 * x12 + y12 * y12 + z12 * z12);
            float d12inv = 1.0f / d12;
            float fc12, fcp12, fa12, fap12, fr12, frp12;

            float d0 = pot_para.ters[D0];
            float a = pot_para.ters[A];
            float r0 = pot_para.ters[R0];
            float s = pot_para.ters[S];
			find_fc_and_fcp
            (
                pot_para.ters[R1], pot_para.ters[R2], 
                pot_para.ters[PI_FACTOR], d12, fc12, fcp12
            );

            find_fa_and_fap(d0, a, r0, s, d12, fa12, fap12);
            find_fr_and_frp(d0, a, r0, s, d12, fr12, frp12);


            // (i,j) part
            float b12 = LDG(g_b, index);
            float factor3 = (frp12 - b12 * fap12) * d12inv;
            float f12x = x12 * factor3 * 0.5f;
            float f12y = y12 * factor3 * 0.5f;
            float f12z = z12 * factor3 * 0.5f;

            // accumulate potential energy
            pot_energy += (fr12 - b12 * fa12) * 0.5f;

            // (i,j,k) part
            float bp12 = LDG(g_bp, index);
            for (int i2 = 0; i2 < neighbor_number; ++i2)
            {
                int index_2 = n1 + number_of_particles * i2;
                int n3 = g_neighbor_list[index_2];
                if (n3 == n2) { continue; }

                float x13 = LDG(g_x, n3) - x1;
                float y13 = LDG(g_y, n3) - y1;
                float z13 = LDG(g_z, n3) - z1;
                dev_apply_mic(h, x13, y13, z13);
                float d13 = sqrt(x13 * x13 + y13 * y13 + z13 * z13);
                float fc13, fa13;
                find_fc
                (
                    pot_para.ters[R1], pot_para.ters[R2], 
                    pot_para.ters[PI_FACTOR], d13, fc13
                );
                find_fa(d0, a, r0, s, d13, fa13);
                float bp13 = LDG(g_bp, index_2);
                float one_over_d12d13 = 1.0f / (d12 * d13);
                float cos123 = (x12*x13 + y12*y13 + z12*z13) * one_over_d12d13;
                float cos123_over_d12d12 = cos123 * d12inv * d12inv;
                float g123, gp123;
                find_g_and_gp
                (
                    pot_para.ters[H], cos123, g123, gp123
                );

                // derivatives with cosine
                float dc = -fc12 * bp12 * fa12 * fc13 * gp123
                            -fc12 * bp13 * fa13 * fc13 * gp123;
                // derivatives with rij
                float dr = -fcp12 * bp13 * fa13 * g123 * fc13 * d12inv;

                float cos_d = x13 * one_over_d12d13 - x12 * cos123_over_d12d12;
                f12x += (x12 * dr + dc * cos_d)*0.5f;
                cos_d = y13 * one_over_d12d13 - y12 * cos123_over_d12d12;
                f12y += (y12 * dr + dc * cos_d)*0.5f;
                cos_d = z13 * one_over_d12d13 - z12 * cos123_over_d12d12;
                f12z += (z12 * dr + dc * cos_d)*0.5f;
            }
            g_f12x[index] = f12x; 
            g_f12y[index] = f12y; 
            g_f12z[index] = f12z;
        }
        // save potential
        g_potential[n1] += pot_energy;
    }
}


static __global__ void find_force_tersoff_step3
(
    int number_of_particles, int *Na, int *Na_sum,
    int *g_neighbor_number, int *g_neighbor_list,
    const float* __restrict__ g_f12x,
    const float* __restrict__ g_f12y,
    const float* __restrict__ g_f12z,
    const float* __restrict__ g_x,
    const float* __restrict__ g_y,
    const float* __restrict__ g_z,
    const float* __restrict__ g_box,
    float *g_fx, float *g_fy, float *g_fz,
    float *g_virial,
	Pot_Para pot_para
)
{
    int N1 = Na_sum[blockIdx.x];
    int N2 = N1 + Na[blockIdx.x];
    int n1 = N1 + threadIdx.x;
    if (n1 < N2)
    {
        float s_fx = 0.0f; // force_x
        float s_fy = 0.0f; // force_y
        float s_fz = 0.0f; // force_z
        float s_virial_xx = 0.0f;
        float s_virial_yy = 0.0f;
        float s_virial_zz = 0.0f;
        float s_virial_xy = 0.0f;
        float s_virial_yz = 0.0f;
        float s_virial_zx = 0.0f;
        const float* __restrict__ h = g_box + 18 * blockIdx.x;
        int neighbor_number = g_neighbor_number[n1];
        float x1 = LDG(g_x, n1); 
        float y1 = LDG(g_y, n1); 
        float z1 = LDG(g_z, n1);
        for (int i1 = 0; i1 < neighbor_number; ++i1)
        {
            int index = i1 * number_of_particles + n1;
            int n2 = g_neighbor_list[index];
            int neighbor_number_2 = g_neighbor_number[n2];
            float x12  = LDG(g_x, n2) - x1;
            float y12  = LDG(g_y, n2) - y1;
            float z12  = LDG(g_z, n2) - z1;
            dev_apply_mic(h, x12, y12, z12);
            float f12x = LDG(g_f12x, index);
            float f12y = LDG(g_f12y, index);
            float f12z = LDG(g_f12z, index);
            int offset = 0;
            for (int k = 0; k < neighbor_number_2; ++k)
            {
                if (n1 == g_neighbor_list[n2 + number_of_particles * k])
                { offset = k; break; }
            }
            index = offset * number_of_particles + n2;
            float f21x = LDG(g_f12x, index);
            float f21y = LDG(g_f12y, index);
            float f21z = LDG(g_f12z, index);

            // per atom force
            s_fx += f12x - f21x; 
            s_fy += f12y - f21y; 
            s_fz += f12z - f21z; 

            // per-atom virial
            s_virial_xx -= x12 * (f12x - f21x) * 0.5f;
            s_virial_yy -= y12 * (f12y - f21y) * 0.5f;
            s_virial_zz -= z12 * (f12z - f21z) * 0.5f;
            s_virial_xy -= x12 * (f12y - f21y) * 0.5f;
            s_virial_yz -= y12 * (f12z - f21z) * 0.5f;
            s_virial_zx -= z12 * (f12x - f21x) * 0.5f;
        }
        // save force
        g_fx[n1] += s_fx * pot_para.ters[LK]; 
        g_fy[n1] += s_fy * pot_para.ters[LK];
        g_fz[n1] += s_fz * pot_para.ters[LK];
        // save virial
        g_virial[n1] += s_virial_xx;
        g_virial[n1 + number_of_particles] += s_virial_yy;
        g_virial[n1 + number_of_particles * 2] += s_virial_zz;
        g_virial[n1 + number_of_particles * 3] += s_virial_xy;
        g_virial[n1 + number_of_particles * 4] += s_virial_yz;
        g_virial[n1 + number_of_particles * 5] += s_virial_zx;
    }
}


void Potential::find_force
(
    int Nc, int N, int *Na, int *Na_sum,
    int max_Na, int *type, float *h, Neighbor *neighbor,
    float *r, float *f, float *virial, float *pe
)
{
	find_force_tersoff_step0<<<Nc, max_Na>>>
    (
        N, Na, Na_sum, neighbor->NN_global, neighbor->NL_global, 
        pot_para, r, r+N, r+N*2, h, f, f+N, f+N*2, virial, pe
    );
    CUDA_CHECK_KERNEL
    find_force_tersoff_step1<<<Nc, max_Na>>>
    (
        N, Na, Na_sum,
        neighbor->NN, neighbor->NL, type, pot_para, r, r+N, r+N*2, h, b, bp
    );
    CUDA_CHECK_KERNEL
    find_force_tersoff_step2<<<Nc, max_Na>>>
    (
        N, Na, Na_sum, neighbor->NN, neighbor->NL, type, 
        pot_para, b, bp, r, r+N, r+N*2, h, pe, f12x, f12y, f12z
    );
    CUDA_CHECK_KERNEL
    find_force_tersoff_step3<<<Nc, max_Na>>>
    (
        N, Na, Na_sum, neighbor->NN, neighbor->NL, 
        f12x, f12y, f12z, r, r+N, r+N*2, h, f, f+N, f+N*2, virial, pot_para
    );
    CUDA_CHECK_KERNEL
}


