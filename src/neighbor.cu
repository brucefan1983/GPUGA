#include "hip/hip_runtime.h"
/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
find the neighbor list
------------------------------------------------------------------------------*/


#include "neighbor.cuh"
#include "mic.cuh"
#include "error.cuh"


Neighbor::~Neighbor(void)
{
    CHECK(hipFree(NN));
    CHECK(hipFree(NL));
	CHECK(hipFree(NN_global));
    CHECK(hipFree(NL_global));
}


static __global__ void gpu_find_neighbor
(
    int N, int *Na, int *Na_sum,
    float cutoff_square, float cutoff_manybody_square, const float* __restrict__ box, 
    int *NN, int *NL, int *NN_global, int *NL_global, float *x, float *y, float *z
)
{
    int N1 = Na_sum[blockIdx.x];
    int N2 = N1 + Na[blockIdx.x];
    int n1 = N1 + threadIdx.x;
    if (n1 < N2)
    {
        const float* __restrict__ h = box + 18 * blockIdx.x;
        float x1 = x[n1];  
        float y1 = y[n1];  
        float z1 = z[n1];
        int count = 0;
		int count_global = 0;
        for (int n2 = N1; n2 < N2; ++n2)
        { 
            if (n2 == n1) { continue; }
            float x12 = x[n2]-x1; 
            float y12 = y[n2]-y1; 
            float z12 = z[n2]-z1;
            dev_apply_mic(h, x12, y12, z12);
            float distance_square = x12 * x12 + y12 * y12 + z12 * z12;
            if (distance_square < cutoff_square && distance_square > cutoff_manybody_square){ NL_global[count_global++ * N + n1] = n2; }		
			if (distance_square < cutoff_manybody_square){ NL[count++ * N + n1] = n2; }
        }
		NN_global[n1] = count_global;
        NN[n1] = count;
    }
}


void Neighbor::compute
(
    int Nc, int N, int max_Na, int *Na, int *Na_sum, float *r, float *h
)
{
    int m1 = sizeof(int) * N;
    CHECK(hipMallocManaged((void**)&NN, m1));
    CHECK(hipMallocManaged((void**)&NL, m1 * max_Na));
	CHECK(hipMallocManaged((void**)&NN_global, m1));
    CHECK(hipMallocManaged((void**)&NL_global, m1 * max_Na));
    float rc2 = cutoff * cutoff;
	float rl2 = cutoff_local * cutoff_local;
    gpu_find_neighbor<<<Nc, max_Na>>>
    (N, Na, Na_sum, rc2, rl2, h, NN, NL, NN_global, NL_global, r, r+N, r+N*2);
    CUDA_CHECK_KERNEL

    CHECK(hipDeviceSynchronize());
    for (int nc = 0; nc < Nc; ++nc)
    {
        printf("NN[%d]=%d,", nc, NN[Na_sum[nc]]);
        if (0 == (nc + 1) % 8) printf("\n");
    }
	for (int nc = 0; nc < Nc; ++nc)
    {
        printf("NN_global[%d]=%d,", nc, NN_global[Na_sum[nc]]);
        if (0 == (nc + 1) % 8) printf("\n");
    }
}


