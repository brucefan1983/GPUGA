/*
    Copyright 2019 Zheyong Fan
    This file is part of GPUGA.
    GPUGA is free software: you can redistribute it and/or modify
    it under the terms of the GNU General Public License as published by
    the Free Software Foundation, either version 3 of the License, or
    (at your option) any later version.
    GPUGA is distributed in the hope that it will be useful,
    but WITHOUT ANY WARRANTY; without even the implied warranty of
    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
    GNU General Public License for more details.
    You should have received a copy of the GNU General Public License
    along with GPUGA.  If not, see <http://www.gnu.org/licenses/>.
*/


/*----------------------------------------------------------------------------80
Some functions for dealing with text files. Written by Mikko Ervasti.
------------------------------------------------------------------------------*/


#include "read_file.cuh"
#include "error.cuh"
#include <errno.h>
#include <ctype.h>


// Read the input file to memory
char* get_file_contents (char *filename)
{
    char *contents;
    int contents_size;
    FILE *in = my_fopen(filename, "r");
    // Find file size
    fseek(in, 0, SEEK_END);
    contents_size = ftell(in);
    rewind(in);
    MY_MALLOC(contents, char, contents_size + 1);
    int size_read_in = fread(contents, sizeof(char), contents_size, in);
    if (size_read_in != contents_size)
    {
        print_error ("File size mismatch.");
    }
    fclose(in);
    contents[contents_size] = '\0'; // Assures proper null termination
    return contents;
}


// Parse a single row
char* row_find_param (char *s, char *param[], int *num_param)
{
    *num_param = 0;
    int start_new_word = 1, comment_found = 0;
    if (s == NULL) return NULL;
    while(*s)
    {
        if(*s == '\n')
        {
            *s = '\0';
            return s + sizeof(char);
        }
        else if (comment_found) { } // Do nothing 
        else if (*s == '#')
        {
            *s = '\0';
            comment_found = 1;
        }
        else if(isspace(*s))
        {
            *s = '\0';
            start_new_word = 1;
        }
        else if (start_new_word)
        {
            param[*num_param] = s;
            ++(*num_param);
            start_new_word = 0;
        }
        ++s;
    }
    return NULL;
}


int is_valid_int (const char *s, int *result)
{
    if (s == NULL || *s == '\0') { return 0; }
    char *p;
    errno = 0;
    *result = (int) strtol (s, &p, 0);
    if (errno != 0 || s == p || *p != 0) { return 0; }
    else {return 1; }
}


int is_valid_real (const char *s, float *result)
{
    if (s == NULL || *s == '\0') { return 0; }
    char *p;
    errno = 0;
    *result = strtod (s, &p);
    if (errno != 0 || s == p || *p != 0) { return 0; }
    else { return 1; }
}


